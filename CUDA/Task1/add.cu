
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;  // Current thread within grid
  int stride = gridDim.x * blockDim.x;                // Number of threads in the grid
  for (int i = index; i < n; i+= stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements. 1 should be left shifted 20 times, ie 2^20

  // Allocate Unified Memory -- accessible from CPU or GPU

  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));    // CUDA equivalent of defining a
  hipMallocManaged(&y, N*sizeof(float));    // pointer of float type with size N

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements across 256 GPU threads
  int blockSize = 256;                               // 256 threads per GPU block on P100
  int numBlocks = (N + blockSize - 1) / blockSize;   // round up the number of blocks

  add<<<numBlocks,blockSize>>>(N, x, y);


  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);    // CUDA equivalent of delete[] x;
  hipFree(y);

  return 0;
}
                      